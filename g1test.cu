#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022-2023 Dag Arne Osvik
// Copyright 2022-2023 Luan Cardoso dos Santos

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include "g1.cuh"
#include "g1test.cuh"

__managed__ testval_t testval[TESTVALS];

////////////////////////////////////////////////////////////

/**
 * @brief initialization
 * 
 */
void init() {

    printf("%s\n", __func__);

    testinit();

    /*
    uint64_t t[2*TESTVALS];

    FILE *pf = fopen("/dev/urandom", "r");

    if (!pf)
        return;

    size_t result = fread(&testval[i], sizeof(testval_t), TESTVALS-i, pf);
    */
}

////////////////////////////////////////////////////////////
//Shorthand for testing a function, with an error check and timer
#define TEST(X) \
    start = clock(); \
    X <<<grid,block>>> (&testval[0]); \
    err = hipDeviceSynchronize(); \
    end = clock(); \
    if (err != hipSuccess) printf("Error %d (%s)\n", err, hipGetErrorName(err)); \
    printf(" (%.2f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

////////////////////////////////////////////////////////////

/**
 * @brief Test for points in G1
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
int main(int argc, char **argv) {
    clock_t start, end;
    hipError_t err;
#if 1
    dim3 block(1,1,1);
    dim3 grid(1,1,1);
#else
    dim3 block(32,8,1);
    dim3 grid(82,1,1);
#endif

    unsigned rows = 2;

    if (argc > 1)
        rows = atoi(argv[1]);

    if (rows < 1)
        rows = 1;

    if (rows > 512)
        rows = 512;

    init();

    TEST(G1TestKAT);
    TEST(G1TestFibonacci);
    TEST(G1TestDbl);

    G1TestFFT(rows);

    return err;
}

// vim: ts=4 et sw=4 si
