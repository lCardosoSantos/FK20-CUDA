#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022 Dag Arne Osvik

#include <stdio.h>

#include "g1.cuh"
#include "fk20.cuh"
#include "fk20test.cuh"
#include "fk20_testvector.cuh"

static __managed__ uint8_t cmp[16*512];
static __managed__ fr_t fr_tmp[16*512];
static __managed__ g1p_t g1p_tmp[512];

void FK20TestFFT() {
    printf(">>>> FFT tests\n");

    toeplitz_coefficients2toeplitz_coefficients_fft(toeplitz_coefficients, toeplitz_coefficients_fft); 
    h2h_fft(h, h_fft);
    h_fft2h(h_fft, h);
    hext_fft2h(hext_fft, h);
    hext_fft2h_fft(hext_fft, h_fft);

}

void toeplitz_coefficients2toeplitz_coefficients_fft(fr_t toeplitz_coefficients_l[16][512], fr_t toeplitz_coefficients_fft_l[16][512]){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    printf("=== RUN   %s\n", "fr_fft: toeplitz_coefficients -> toeplitz_coefficients_fft");
    memset(fr_tmp, 0xdeadbeef,16*512*sizeof(fr_t)); //pattern on tmp dest.
    start = clock();
    fr_fft_wrapper<<<16, 256, fr_sharedmem>>>(fr_tmp, (fr_t *)toeplitz_coefficients_l);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fr_fft_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<16*512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "fr_eq_wrapper", cmp, 512, fr_tmp, h_fft); fflush(stdout);

    fr_eq_wrapper<<<256, 32>>>(cmp, 16*512, fr_tmp, (fr_t *)toeplitz_coefficients_fft_l);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error fr_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check FFT result

    for (int i=0; pass && i<16*512; i++)
        if (cmp[i] != 1) {
            printf("FFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);
    //////////////////////////////////////////////////
}

void h2h_fft(g1p_t h_l[512], g1p_t h_fft_l[512]){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_fft_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "g1p_fft: h -> h_fft");
    memset(g1p_tmp,0xdeadbeef,512*sizeof(g1p_t)); //pattern on tmp dest

    start = clock();
    g1p_fft_wrapper<<<1, 256, g1p_sharedmem>>>(g1p_tmp, h_l);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess) printf("Error g1p_fft_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h_fft); fflush(stdout);

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512, g1p_tmp, h_fft_l);

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Check FFT result

    for (int i=0; pass && i<512; i++)
        if (cmp[i] != 1) {
            printf("FFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}

void h_fft2h(g1p_t h_fft_l[512], g1p_t h_l[512]){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_ift_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "g1p_ift: h_fft -> h");
    memset(g1p_tmp,0xdeadbeef,512*sizeof(g1p_t)); //pattern on tmp dest
    start = clock();
    g1p_ift_wrapper<<<1, 256, g1p_sharedmem>>>(g1p_tmp, h_fft_l);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error g1p_ift_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h); fflush(stdout);

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512, g1p_tmp, h_l);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check IFT result

    for (int i=0; pass && i<512; i++)
        if (cmp[i] != 1) {
            printf("IFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}

void hext_fft2h(g1p_t hext_fft_l[512], g1p_t h_l[512]){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_ift_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "g1p_ift: hext_fft -> h");
    memset(g1p_tmp,0xdeadbeef,512*sizeof(g1p_t)); //pattern on tmp dest
    start = clock();
    g1p_ift_wrapper<<<1, 256, g1p_sharedmem>>>(g1p_tmp, hext_fft_l);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error g1p_ift_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h); fflush(stdout);

    g1p_eq_wrapper<<<8, 32>>>(cmp, 256, g1p_tmp, h_l);    // Note: h, not hext, hence 256, not 512

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check IFT result

    for (int i=0; pass && i<256; i++)
        if (cmp[i] != 1) {
            printf("IFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}


// vim: ts=4 et sw=4 si
void hext_fft2h_fft(g1p_t hext_fft_l[512], g1p_t h_fft_l[512]){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(fk20_hext_fft2h_fft), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "fk20_hext_fft2h_fft: hext_fft -> h_fft");
    memset(g1p_tmp,0xdeadbeef,512*sizeof(g1p_t)); //pattern on tmp dest

    start = clock();
    fk20_hext_fft2h_fft<<<1, 256, g1p_sharedmem>>>(g1p_tmp, hext_fft_l);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess) printf("Error fk20_hext_fft2h_fft: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h_fft); fflush(stdout);

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512, g1p_tmp, h_fft_l);

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Check FFT result

    for (int i=0; pass && i<512; i++)
        if (cmp[i] != 1) {
            printf("FFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}