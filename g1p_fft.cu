#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022-2023 Dag Arne Osvik
// Copyright 2022-2023 Luan Cardoso dos Santos

#include <stdio.h>

#include "g1.cuh"
#include "fk20.cuh"

// Workspace in shared memory

extern __shared__ g1p_t g1p_tmp[];

/**
 * @brief FFT of size 512 over G1 with projective coordinates. Input and output
 * arrays may overlap.
 * This function must be called with 256 threads per block, i.e. dim3(256,1,1).
 * No interleaving of data for different FFTs
 * 
 * @param[out] output pointer to 512 elements array of g1p_t
 * @param[in] input pointer to 512 elements array of g1p_t
 * @return void
 */
__device__ void g1p_fft(g1p_t *output, const g1p_t *input) {

    if (gridDim.y  !=   1) return;
    if (gridDim.z  !=   1) return;
    if (blockDim.x != 256) return;
    if (blockDim.y !=   1) return;
    if (blockDim.z !=   1) return;

    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number
    unsigned l, r, w, src, dst;

    // Adjust IO pointers to point at each thread block's data

    input  += 512*bid;
    output += 512*bid;

    // Copy inputs to workspace

    src = tid;
    // dst = 9 last bits of src reversed
    asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(dst) : "r"(src << (32-9)));

    g1p_cpy(g1p_tmp[dst], input[src]);

    src |= 256;
    dst |= 1;

    g1p_cpy(g1p_tmp[dst], input[src]);

    __syncthreads();

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    //g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncwarp();

    //// Stage 1

    w = (tid & 1) << 7;
    l = tid + (tid & -2U);
    r = l | 2;

    if (w) g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncwarp();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncwarp();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncwarp();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;

    g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncwarp();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncthreads();

    //// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    g1p_mul(g1p_tmp[r], fr_roots[w]);
    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);

    __syncthreads();

    // Copy results to output, no shuffle

    src = tid;
    dst = src;

    g1p_cpy(output[dst], g1p_tmp[src]);

    src += 256;
    dst += 256;

    g1p_cpy(output[dst], g1p_tmp[src]);
}

/**
 * @brief Inverse FFT of size 512 over G1 with projective coordinates. Input and output
 * arrays may overlap.
 * This function must be called with 256 threads per block, i.e. dim3(256,1,1).
 * No interleaving of data for different FFTs
 * 
 * @param[out] output pointer to 512 elements array of g1p_t
 * @param[in] input pointer to 512 elements array of g1p_t
 * @return void
 */
__device__ void g1p_ift(g1p_t *output, const g1p_t *input) {

    if (gridDim.y  !=   1) return;
    if (gridDim.z  !=   1) return;
    if (blockDim.x != 256) return;
    if (blockDim.y !=   1) return;
    if (blockDim.z !=   1) return;

    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number
    unsigned l, r, w, src, dst;

    // Adjust IO pointers to point at each thread block's data

    input  += 512*bid;
    output += 512*bid;

    // Copy inputs to workspace, no shuffle

    src = tid;
    dst = src;

    g1p_cpy(g1p_tmp[dst], input[src]);

    src += 256;
    dst += 256;

    g1p_cpy(g1p_tmp[dst], input[src]);

    __syncthreads();

    //// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncwarp();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncwarp();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncwarp();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncwarp();

    //// Stage 1

    w = (tid & 1) << 0;
    l = tid + (tid & -2U);
    r = l | 2;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    g1p_mul(g1p_tmp[l], fr_roots[513]);    // 2**-9
    g1p_mul(g1p_tmp[r], fr_roots[513+w]);  // w ? 2**-9/fr_roots[128] : 2**-9

    __syncwarp();

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    g1p_addsub(g1p_tmp[l], g1p_tmp[r]);
    //g1p_mul(g1p_tmp[r], fr_roots[512-w]);

    __syncthreads();

    // Copy results to output

    dst = tid;
    // src = 9 last bits of dst reversed
    asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(src) : "r"(dst << (32-9)));

    g1p_cpy(output[dst], g1p_tmp[src]);

    dst |= 256;
    src |= 1;

    g1p_cpy(output[dst], g1p_tmp[src]);
}

// Kernel wrappers for device-side FFT functions

/**
 * @brief wrapper for g1p_fft: FFT for arrays of g1p_t with length 512
 * 
 * Executes an FFT over many arrays of arrays of g1p_t with length 512. One array per block. input and 
 * output can overlap without side effects. There is no interleaving of data for
 * different FFTs.
 * 
 * @param[out] output pointer to 512*blocksize elements array of g1p_t
 * @param[in] input pointer to 512*blocksize elements array of g1p_t
 * @return void 
 */
__global__ void g1p_fft_wrapper(g1p_t *output, const g1p_t *input) { g1p_fft(output, input); }

/**
 * @brief wrapper for g1p_ift: inverse FFT for arrays of g1p_t with length 512
 * 
 * Executes an Inverse FFT over many arrays of arrays of g1p_t with length 512. One array per block. input and 
 * output can overlap without side effects. There is no interleaving of data for
 * different FFTs.
 * 
 * @param[out] output pointer to 512*blocksize elements array of g1p_t
 * @param[in] input pointer to 512*blocksize elements array of g1p_t
 * @return void 
 */
__global__ void g1p_ift_wrapper(g1p_t *output, const g1p_t *input) { g1p_ift(output, input); }

// vim: ts=4 et sw=4 si
