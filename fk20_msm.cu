#include "hip/hip_runtime.h"
#include "fr.cuh"
#include "g1.cuh"
#include "fk20.cuh"

////////////////////////////////////////////////////////////////////////////////

// fk20_msm(): toeplitz_coefficients_fft + xext_fft -> hext_fft

// parameters: 
// - in  xe_fft    array with dimensions [16][512]
// - in  tc_fft    array with dimensions [16][512]
// - out he_fft    array with dimensions [512]

__global__ void fk20_msm(g1p_t *he_fft, const fr_t *tc_fft, const g1p_t *xe_fft) {
    if (gridDim.y  !=   1) return;
    if (gridDim.z  !=   1) return;
    if (blockDim.x != 256) return;  // k
    if (blockDim.y !=   1) return;
    if (blockDim.z !=   1) return;

    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number

    g1p_t a0, a1, t;

    g1p_inf(a0);
    g1p_inf(a1);

    // move pointer for blocks
    he_fft += 512*bid;
    tc_fft += 16*512*bid;

    // MSM Loop
    for (int i=0; i<16; i++) {

        // Multiply and accumulate

        g1p_cpy(t, xe_fft[512*i+tid+0]);
        g1p_mul(t, tc_fft[512*i+tid+0]);
        g1p_add(a0, t);

        g1p_cpy(t, xe_fft[512*i+tid+256]);
        g1p_mul(t, tc_fft[512*i+tid+256]);
        g1p_add(a1, t);
    }

    // hext_fft = a0||a1
    // Store accumulators
    g1p_cpy(he_fft[tid+  0], a0);
    g1p_cpy(he_fft[tid+256], a1);
}

// vim: ts=4 et sw=4 si
