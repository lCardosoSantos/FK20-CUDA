#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022 Dag Arne Osvik

#include "fp.cuh"
#include "fptest.cuh"

#define ITERATIONS 100000

__global__ void FpTestFibonacci(testval_t *) {

    printf("=== RUN   %s\n", __func__);

    bool    pass    = true;
    size_t  count   = 0;

    fp_t x, y, t, u;

    fp_one(x);
    fp_one(y);

    for (int i=0; i<ITERATIONS; i++) {

        fp_cpy(t, x);
        fp_add(x, x, y);

        fp_cpy(u, x);
        fp_sub(u, u, t);

        if (fp_neq(u, y)) {
            printf("x ="); fp_print(x);
            printf("y ="); fp_print(y);
            printf("x+y ="); fp_print(t);
            printf("x+y-x ="); fp_print(u);
            pass = false;
            break;
        }

        ++count;

        fp_cpy(t, y);
        fp_add(y, y, x);

        fp_cpy(u, y);
        fp_sub(u, u, t);

        if (fp_neq(u, x)) {
            printf("x ="); fp_print(x);
            printf("y ="); fp_print(y);
            printf("x+y ="); fp_print(t);
            printf("x+y-y ="); fp_print(u);
            pass = false;
            break;
        }

        ++count;
    }

    for (int i=0; i<ITERATIONS; i++) {
        fp_sub(y, y, x);
        fp_sub(x, x, y);
    }

    if (!fp_isone(x) || !fp_isone(y)) {
        printf("Reverse iteration failed\n");
        printf("x ="); fp_print(x);
        printf("y ="); fp_print(y);
        pass = false;
    }
    else
        ++count;

    printf("%ld tests passed\n", count);

    PRINTPASS(pass);
}

// vim: ts=4 et sw=4 si
