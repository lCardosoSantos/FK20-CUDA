#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022 Dag Arne Osvik

#include "fp.cuh"
#include "fptest.cuh"

__global__ void FpTestSub(testval_t *testval) {

    printf("=== RUN   %s\n", __func__);

    bool    pass    = true;

    unsigned tid = 0;   tid += blockIdx.z;
    tid *= gridDim.y;   tid += blockIdx.y;
    tid *= gridDim.x;   tid += blockIdx.x;
    tid *= blockDim.z;  tid += threadIdx.z;
    tid *= blockDim.y;  tid += threadIdx.y;
    tid *= blockDim.x;  tid += threadIdx.x;

    uint64_t t[6] = { 1L<<33, 0, 0, 0, 0, 0 };
    uint64_t u[6] = { tid, 0, 0, 0, 0, 0 };

    fp_t x, y, z;

    fp_fromUint64(x, t);    // x = 2**33
    fp_fromUint64(y, u);    // y = tid

    if (x[0] != 1L<<33) pass = false;
    if (x[1] !=      0) pass = false;
    if (x[2] !=      0) pass = false;
    if (x[3] !=      0) pass = false;
    if (x[4] !=      0) pass = false;
    if (x[5] !=      0) pass = false;

    if (y[0] != tid) pass = false;
    if (y[1] !=   0) pass = false;
    if (y[2] !=   0) pass = false;
    if (y[3] !=   0) pass = false;
    if (y[4] !=   0) pass = false;
    if (y[5] !=   0) pass = false;

    if (!pass) {
        printf("%d: FAILED after fp_fromUint64\n", tid);
        goto done;
    }

    fp_sub(z, x, y);    // z = x - y

    fp_reduce6(z);

    fp_reduce6(z);

    if (z[0] != (1L<<33)-tid) pass = false;
    if (z[1] !=            0) pass = false;
    if (z[2] !=            0) pass = false;
    if (z[3] !=            0) pass = false;
    if (z[4] !=            0) pass = false;
    if (z[5] !=            0) pass = false;

    if (!pass) {
        printf("%d: FAILED after fp_sub\n", tid);
        printf("0x%016lx%016lx%016lx%016lx%016lx%016lx\n", z[5], z[4], z[3], z[2], z[1], z[0]);
        goto done;
    }

done:
    printf("--- %s: %s\n", pass ? "PASS" : "FAIL", __func__);
}

// vim: ts=4 et sw=4 si
