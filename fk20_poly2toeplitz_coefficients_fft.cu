#include "hip/hip_runtime.h"
#include <stdio.h>

#include "fr.cuh"
#include "fk20.cuh"

static __device__ fr_t fr_tmp[512*16*512];     // 256 KiB memory per threadblock

////////////////////////////////////////////////////////////////////////////////

// fk20_poly2toeplitz_coefficients_fft(): polynomial -> toeplitz_coefficients_fft

// parameters:
// - in  polynomial                 array with 16*512*gridDim.x elements
// - out toeplitz_coefficients_fft  array with 16*512*gridDim.x elements

__global__ void fk20_poly2toeplitz_coefficients_fft(fr_t *toeplitz_coefficients_fft, const fr_t *polynomial) {

    // gridDim.x is the number of rows
    if (gridDim.y  !=   1) return;
    if (gridDim.z  !=   1) return;
    if (blockDim.x != 256) return;  // k
    if (blockDim.y !=   1) return;
    if (blockDim.z !=   1) return;

    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number

    // Accumulators and temporaries in registers or local
    // (thread-interleaved global) memory

    polynomial += 4096 * bid;
    toeplitz_coefficients_fft += 8192 * bid;

    fr_t *fr = fr_tmp + 8192 * bid;

    __syncthreads();

    // Loop

    for (int i=0; i<16; i++) {

        // Copy from the polynomial into half of the coefficient array

        unsigned src = tid*16+15-i;
        unsigned dst = (tid+257)%512;

        if (tid > 0)
            fr_cpy(fr[dst], polynomial[src]);
        else
            fr_zero(fr[dst]);

        __syncthreads();

        // Zero the other half of coefficients before FFT

        fr_zero(fr[tid+1]);

        __syncthreads();

        // Compute FFT

        fr_fft(fr, fr);

        __syncthreads();

        fr_cpy(toeplitz_coefficients_fft[tid], fr[tid]);
        fr_cpy(toeplitz_coefficients_fft[tid+256], fr[tid+256]);

        toeplitz_coefficients_fft += 512;
    }
}

// vim: ts=4 et sw=4 si
