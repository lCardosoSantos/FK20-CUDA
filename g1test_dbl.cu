#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022-2023 Dag Arne Osvik
// Copyright 2022-2023 Luan Cardoso dos Santos

#include <stdio.h>

#include "g1.cuh"
#include "fr.cuh"
#include "g1test.cuh"
#include "g1p_ptx.cuh"

/**
 * @brief Test for point doubling in G1:
 * 
 * p+p==dbl(p)
 * 
 * @return void 
 */
__global__ void G1TestDbl(testval_t *) {
    #define g1p_dbl(q) g1m(OP_DBL, q, q, q, q);
    if ((blockIdx.x | blockIdx.y | blockIdx.z | threadIdx.x | threadIdx.y | threadIdx.z) == 0)
        printf("=== RUN   %s\n", __func__);

    bool    pass    = true;
    size_t  count   = 0;

    g1p_t p, q, u, v;

    g1p_gen(p); // p  = G
    g1p_gen(q); // q  = G

    for (int i=0; pass && i<20000; i++) {

        g1p_cpy(u, p);
        g1p_cpy(v, q);

        g1p_add(p, p);  // p += p
        g1p_dbl(q);     // q *= 2

        if (g1p_neq(p, q)) {
            pass = false;

            printf("%d: FAILED\n", i);
            g1p_print("u   = ", u);
            g1p_print("v   = ", v);
            g1p_print("u+u = ", p);
            g1p_print("2v  = ", q);
        }
        ++count;
    }

    if (!pass || (blockIdx.x | blockIdx.y | blockIdx.z | threadIdx.x | threadIdx.y | threadIdx.z) == 0)
    {
        printf("%ld tests\n", count);

        PRINTPASS(pass);
    }
     #undef g1p_dbl
}


/**
 * @brief Test for point doubling in G1:
 * previous implementation, without unrolling of the PTX to in-register.
 * p+p==dbl(p)
 * 
 * @return void 
 */
__global__ void G1TestDbl_noPTX(testval_t *) {

    if ((blockIdx.x | blockIdx.y | blockIdx.z | threadIdx.x | threadIdx.y | threadIdx.z) == 0)
        printf("=== RUN   %s\n", __func__);

    bool    pass    = true;
    size_t  count   = 0;

    g1p_t p, q, u, v;

    g1p_gen(p); // p  = G
    g1p_gen(q); // q  = G

    for (int i=0; pass && i<20000; i++) {

        g1p_cpy(u, p);
        g1p_cpy(v, q);

        g1p_add(p, p);  // p += p
        g1p_dbl(q);     // q *= 2

        if (g1p_neq(p, q)) {
            pass = false;

            printf("%d: FAILED\n", i);
            g1p_print("u   = ", u);
            g1p_print("v   = ", v);
            g1p_print("u+u = ", p);
            g1p_print("2v  = ", q);
        }
        ++count;
    }

    if (!pass || (blockIdx.x | blockIdx.y | blockIdx.z | threadIdx.x | threadIdx.y | threadIdx.z) == 0)
    {
        printf("%ld tests\n", count);

        PRINTPASS(pass);
    }
}

// vim: ts=4 et sw=4 si
