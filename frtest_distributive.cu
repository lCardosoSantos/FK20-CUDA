#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022-2023 Dag Arne Osvik
// Copyright 2022-2023 Luan Cardoso dos Santos

#include "fr.cuh"
#include "frtest.cuh"


/**
 * @brief Check the distributive property (left of addition):
 * 
 * a(b+c) = ab+ac
 * 
 * @param testval 
 * @return void 
 */
__global__ void FrTestAddDistributiveLeft(testval_t *testval) {

    printf("=== RUN   %s\n", __func__);

    bool    pass    = true;
    size_t  count   = 0;
    fr_t    a, b, c, u, v, w;

    for (int i=0; i<TESTVALS; i++) {
        fr_cpy(a, testval[i]);

        for (int j=0; j<TESTVALS; j++) {
            fr_cpy(b, testval[j]);

            for (int k=j; k<TESTVALS; k++) {
                fr_cpy(c, testval[k]);

                fr_cpy(u, a);
                fr_mul(u, b);   // ab

                fr_cpy(v, a);
                fr_mul(v, c);   // ac

                fr_add(u, v);   // ab+ac

                fr_cpy(v, a);
                fr_cpy(w, b);
                fr_add(w, c);   // b+c
                fr_mul(v, w);   // a(b+c)

                if (fr_neq(u, v)) {
                    pass = false;

                    printf("%d,%d: FAILED: inconsistent result\n", i, j);
                    fr_print("a = ",  testval[i]);
                    fr_print("b = ",  testval[j]);
                    fr_print("c = ",  testval[k]);
                    fr_print("ab+ac = ",  u);
                    fr_print("a(b+c) = ",  v);
                }
                ++count;
            }
        }
    }
    printf("%ld tests\n", count);

    PRINTPASS(pass);
}

/**
 * @brief Check the distributive property (right of addition):
 * 
 * (a+b)c = ac+bc
 * 
 * @param testval 
 * @return void 
 */
__global__ void FrTestAddDistributiveRight(testval_t *testval) {

    printf("=== RUN   %s\n", __func__);

    bool    pass    = true;
    size_t  count   = 0;
    fr_t    a, b, c, u, v;

    for (int i=0; i<TESTVALS; i++) {
        fr_cpy(a, testval[i]);

        for (int j=i; j<TESTVALS; j++) {
            fr_cpy(b, testval[j]);

            for (int k=0; k<TESTVALS; k++) {
                fr_cpy(c, testval[k]);

                fr_cpy(u, a);
                fr_mul(u, c);   // ac

                fr_cpy(v, b);
                fr_mul(v, c);   // bc

                fr_add(u, v);   // ac+bc

                fr_cpy(v, a);
                fr_add(v, b);   // a+b
                fr_mul(v, c);   // (a+b)c

                if (fr_neq(u, v)) {
                    pass = false;

                    printf("%d,%d: FAILED: inconsistent result\n", i, j);
                    fr_print("a = ",  testval[i]);
                    fr_print("b = ",  testval[j]);
                    fr_print("c = ",  testval[k]);
                    fr_print("ac+bc = ",  u);
                    fr_print("(a+b)c = ",  v);
                }
                ++count;
            }
        }
    }
    printf("%ld tests\n", count);

    printf("--- %s: %s\n", pass ? "PASS" : "FAIL", __func__);
}


/**
 * @brief Check the distributive property (left of subtraction):
 * 
 * a(b-c) = ab-ac
 * 
 * @param testval 
 * @return void 
 */
__global__ void FrTestSubDistributiveLeft(testval_t *testval) {

    printf("=== RUN   %s\n", __func__);

    bool    pass    = true;
    size_t  count   = 0;
    fr_t    a, b, c, u, v, w;

    for (int i=0; i<TESTVALS; i++) {
        fr_cpy(a, testval[i]);

        for (int j=0; j<TESTVALS; j++) {
            fr_cpy(b, testval[j]);

            for (int k=0; k<TESTVALS; k++) {
                fr_cpy(c, testval[k]);

                fr_cpy(u, a);
                fr_mul(u, b);   // ab

                fr_cpy(v, a);
                fr_mul(v, c);   // ac

                fr_sub(u, v);   // ab-ac

                fr_cpy(v, a);
                fr_cpy(w, b);
                fr_sub(w, c);   // b-c
                fr_mul(v, w);   // a(b-c)

                if (fr_neq(u, v)) {
                    pass = false;

                    printf("%d,%d: FAILED: inconsistent result\n", i, j);
                    fr_print("a = ",  testval[i]);
                    fr_print("b = ",  testval[j]);
                    fr_print("c = ",  testval[k]);
                    fr_print("ab-ac = ",  u);
                    fr_print("a(b-c) = ",  v);
                }
                ++count;
            }
        }
    }
    printf("%ld tests\n", count);

    printf("--- %s: %s\n", pass ? "PASS" : "FAIL", __func__);
}

// (a-b)c = ac-bc

/**
 * @brief Check the distributive property (right of subtraction):
 * 
 * (a-b)c = ac-bc
 * 
 * @param testval 
 * @return void 
 */
__global__ void FrTestSubDistributiveRight(testval_t *testval) {

    printf("=== RUN   %s\n", __func__);

    bool    pass    = true;
    size_t  count   = 0;
    fr_t    a, b, c, u, v;

    for (int i=0; i<TESTVALS; i++) {
        fr_cpy(a, testval[i]);

        for (int j=0; j<TESTVALS; j++) {
            fr_cpy(b, testval[j]);

            for (int k=0; k<TESTVALS; k++) {
                fr_cpy(c, testval[k]);

                fr_cpy(u, a);
                fr_mul(u, c);   // ac

                fr_cpy(v, b);
                fr_mul(v, c);   // bc

                fr_sub(u, v);   // ac-bc

                fr_cpy(v, a);
                fr_sub(v, b);   // a-b
                fr_mul(v, c);   // (a-b)c

                if (fr_neq(u, v)) {
                    pass = false;

                    printf("%d,%d: FAILED: inconsistent result\n", i, j);
                    fr_print("a = ",  testval[i]);
                    fr_print("b = ",  testval[j]);
                    fr_print("c = ",  testval[k]);
                    fr_print("ac-bc = ",  u);
                    fr_print("(a-b)c = ",  v);
                }
                ++count;
            }
        }
    }
    printf("%ld tests\n", count);

    printf("--- %s: %s\n", pass ? "PASS" : "FAIL", __func__);
}

// vim: ts=4 et sw=4 si
