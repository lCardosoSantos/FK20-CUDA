#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022-2023 Dag Arne Osvik
// Copyright 2022-2023 Luan Cardoso dos Santos
//#define SHAREDMEMROOTS

#include <cassert>
#include <cstdio>

#include "g1.cuh"
#include "fk20.cuh"

// Workspace in shared mem
extern __shared__ g1p_t g1p_tmp[]; //512 * 3 * 6 * 8; // 512 points * 3 residues/point * 6 words/residue * 8 bytes/word 
                                   //= 72 KiB
#ifdef SHAREDMEMROOTS
__device__ fr_t *fr_roots_sm;
#endif

__device__ void fft();
__device__ void ift();
__device__ void wsm_g1p(unsigned index, const g1p_t *input);
__device__ void rsm_g1p(unsigned index, g1p_t *output);
__device__ void ism_g1p(unsigned index);
__device__ void wsm_fr(unsigned index, fr_t &input);
__device__ void rsm_fr(unsigned index, fr_t &output);

__global__ void fk20_hext_fft2h_fft(g1p_t *h_fft, const g1p_t *hext_fft){
    assert(gridDim.y  ==   1);
    assert(gridDim.z  ==   1);
    assert(blockDim.x == 256);
    assert(blockDim.y ==   1);
    assert(blockDim.z ==   1);

    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number
    unsigned secondHalfIndexes = tid;

    //Move pointer to block
    hext_fft  += 512*bid;
    h_fft += 512*bid;

#ifdef SHAREDMEMROOTS
    //Move pointer of Fr Sharedmem
    fr_roots_sm = (fr_t*)&g1p_tmp[512];
    //write into sharedmem
    wsm_fr(tid    , fr_roots[tid]);
    wsm_fr(tid+256, fr_roots[tid+256]);
    if(tid<3) wsm_fr(tid+512, fr_roots[tid+512]);
#endif

    // STEP1 hext_fft -> hext 
    wsm_g1p(tid, &hext_fft[tid]);
    wsm_g1p(tid+256, &hext_fft[tid+256]);

    // IFT of sharedmem
    ift();

    // STEP2 hext -> h
    // Zeroing second half. IFT did not reorder the array in shared mem on the last step
    asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(secondHalfIndexes) : "r"(tid << (32-9)));
    secondHalfIndexes |= 1;
    ism_g1p(secondHalfIndexes);

    // STEP3 h -> h_fft
    // FFT of sharedmem
    fft();
    //move from shared mem into h_fft
    rsm_g1p(tid, &h_fft[tid]);
    rsm_g1p(tid+256, &h_fft[tid+256]);
}

#ifndef SHAREDMEMROOTS
__device__ void ift(){
    unsigned l, r, w, tid;
    tid = threadIdx.x;

    g1p_t gl, gr;
//// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); 
    g1p_addsub(gl, gr);
    g1p_mul(gr, fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    g1p_mul(gr, fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    g1p_mul(gr, fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    g1p_mul(gr, fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;
    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    g1p_mul(gr, fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    g1p_mul(gr, fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    g1p_mul(gr, fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 1

    w = (tid & 1) << 0;
    l = tid + (tid & -2U);
    r = l | 2;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    g1p_mul(gl, fr_roots[513]);    // 2**-9
    g1p_mul(gr, fr_roots[513+w]);  // w ? 2**-9/fr_roots[128] : 2**-9
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_addsub(gl, gr);
    //g1p_mul(output[r], fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //last move not needed, everything in sharedmem
    // Last move
    //dst = threadIdx.x;
    // src = 9 last bits of dst reversed
    //asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(src) : "r"(dst << (32-9)));

    // if (threadIdx.x == 0){
    //     unsigned SRC, DST;
    //     printf(">>> maping at end of ift\n");
    //     for(int i=0; i<256; i++){
    //         DST=i;
    //         asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(SRC) : "r"(DST << (32-9)));
    //         printf("%3d ift:(%3u -> %3u) ", i, SRC, DST);

    //         DST|=256;
    //         SRC|=1;
    //         printf("(%3u -> %3u) ", SRC, DST);

    //         SRC = i; 
    //         asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(DST) : "r"(SRC << (32-9)));
    //         printf("fft: (%3u -> %3u) ", SRC, DST);
    //         SRC |= 256;
    //         DST |= 1; 
    //         printf("(%3u -> %3u)\n", SRC, DST);
    //     }
    // }
}

__device__ void fft(){
    // unsigned src, dst;
    unsigned l, r, w, tid;
    g1p_t gl, gr;

    tid = threadIdx.x;
    // src = threadIdx.x;

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    //g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 1

    w = (tid & 1) << 7;
    l = tid + (tid & -2U);
    r = l | 2;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    if (w) g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr);
    g1p_mul(gr, fr_roots[w]);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

}

#else
__device__ void ift(){
    unsigned src, dst;
    unsigned l, r, w, tid;
    tid = threadIdx.x;
    src = threadIdx.x;
    dst = src;

    g1p_t gl, gr;
    fr_t roots;
//// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gr, roots);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gr, roots);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gr, roots);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gr, roots);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;
    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gr, roots);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gr, roots);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gr, roots);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 1

    w = (tid & 1) << 0;
    l = tid + (tid & -2U);
    r = l | 2;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(513, roots);
    g1p_addsub(gl, gr);
    g1p_mul(gl, roots);    // 2**-9
    rsm_fr(513+w, roots);
    g1p_mul(gr, roots);  // w ? 2**-9/fr_roots[128] : 2**-9
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); //rsm_fr(513, roots);
    g1p_addsub(gl, gr);
    //g1p_mul(output[r], fr_roots[512-w]);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //last move not needed, everything in sharedmem
    // Last move
    //dst = threadIdx.x;
    // src = 9 last bits of dst reversed
    //asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(src) : "r"(dst << (32-9)));

    // if (threadIdx.x == 0){
    //     unsigned SRC, DST;
    //     printf(">>> maping at end of ift\n");
    //     for(int i=0; i<256; i++){
    //         DST=i;
    //         asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(SRC) : "r"(DST << (32-9)));
    //         printf("%3d ift:(%3u -> %3u) ", i, SRC, DST);

    //         DST|=256;
    //         SRC|=1;
    //         printf("(%3u -> %3u) ", SRC, DST);

    //         SRC = i; 
    //         asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(DST) : "r"(SRC << (32-9)));
    //         printf("fft: (%3u -> %3u) ", SRC, DST);
    //         SRC |= 256;
    //         DST |= 1; 
    //         printf("(%3u -> %3u)\n", SRC, DST);
    //     }
    // }
}

__device__ void fft(){
    // unsigned src, dst;
    unsigned l, r, w, tid;
    g1p_t gl, gr;
    fr_t roots;

    tid = threadIdx.x;
    // src = threadIdx.x;

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    //g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 1

    w = (tid & 1) << 7;
    l = tid + (tid & -2U);
    r = l | 2;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    if (w) g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    // __syncthreads();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

    __syncthreads();

    //// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    rsm_g1p(l, &gl); rsm_g1p(r, &gr); rsm_fr(512-w, roots);
    g1p_mul(gr, roots);
    g1p_addsub(gl, gr);
    wsm_g1p(l, &gl); wsm_g1p(r, &gr);

}
#endif


// Set g1p_tmp[index] to infinity, with staggered write.
__device__ void ism_g1p(unsigned index){
    uint32_t *bankPointer;
    bankPointer = (uint32_t*) g1p_tmp;
    int smi; //index of the shared memory (as uint32_t)

    for(int widx=0; widx<sizeof(g1p_t)/4; widx++){ //32 words
        smi = (index/32)*32*36; // puts the index bank[0][0] for this warp
        smi += index%32;        // moves the index foward to the correct bank
        smi += widx*32;         // moves the index inside the bank for the correct word

        bankPointer[smi] = 0x00000000;
    }

    smi = (index/32)*32*36; // puts the index bank[0][0] for this warp
    smi += index%32;        // moves the index foward to the correct bank
    smi += 12*32;           // moves the index inside the bank to DWORD12 == g1p_tmp.y[0]
    bankPointer[smi] = 0x00000001;
}

// Staggered Write on Shared Memory
__device__ void wsm_g1p(unsigned index, const g1p_t *input){
    //writes input in the sharedmem, staggering such that threadIdx.x
    //will writer to memorybank threadIdx.x%32
    //Index is a value that maps 512 g1p_t values into sharedmem.
    //There are 32 SM banks, 32bit wide. Each of the 32 Threads in the WARP 
    //should read from a different sharedmem banks for optimal performance.

    //considers that shared mem pointer is called g1p_tmp
    uint32_t *bankPointer;
    uint32_t *g1Pointer;
    bankPointer = (uint32_t*) g1p_tmp;
    g1Pointer=(uint32_t*) input;

    int smi; //index of the shared memory (as uint32_t)

    for(int widx=0; widx<sizeof(g1p_t)/4; widx++){ //32 words
        smi = (index/32)*32*36; // puts the index bank[0][0] for this warp
        smi += index%32;        // moves the index foward to the correct bank
        smi += widx*32;         // moves the index inside the bank for the correct word

        bankPointer[smi] = g1Pointer[widx];
        //if (index ==0) printf("tw:%d idx:%d %d=%d\n", threadIdx.x, index, smi,widx);
    }

}

// Staggered Read from Shared Memory
__device__ void rsm_g1p(unsigned index, g1p_t *output){
    //reads the scathered value on sharedmem and write into 
    //output.
    //There are 32 SM banks, 32bit wide. Each of the 32 Threads in the WARP 
    //should read from a different sharedmem banks for optimal performance.

    //considers that shared mem pointer is called g1p_tmp

    uint32_t *bankPointer;
    uint32_t *g1Pointer;
    bankPointer = (uint32_t*) g1p_tmp;
    g1Pointer=(uint32_t*) output;

    int smi; //shared memory index (as uint32_t)

    for(int widx=0; widx<sizeof(g1p_t)/4; widx++){ //32 words
        smi = (index/32)*32*36; // puts the index bank[0][0] for this warp (32 banks 32bits wide, 36words in g1p_t)
        smi += index%32;        // moves the index foward to the correct bank
        smi += widx*32;         // moves the index inside the bank for the correct word

        g1Pointer[widx]  = bankPointer[smi];
        //printf("tr:%d idx:%d %d=%d\n", threadIdx.x, index, widx, smi);
    }

    return;
}


#ifdef SHAREDMEMROOTS

__device__ void wsm_fr(unsigned index, fr_t &input){
    //considers that shared mem pointer is called fr_roots_sm
    uint32_t *bankPointer;
    uint32_t *frPointer;
    bankPointer = (uint32_t*) fr_roots_sm;
    frPointer=(uint32_t*) input;

    //index changing: similar to aes shiftcolumns, on 32 columns
    unsigned rotN = index/32; //rotation amount
    index = ((index + 32 - rotN)%32)+ (rotN*32);

    int smi; //index of the shared memory (as uint32_t)

    for(int widx=0; widx<sizeof(fr_t)/4; widx++){ //8 words
        smi = (index/32)*32*8; // puts the index bank[0][0] for this warp
        smi += index%32;        // moves the index foward to the correct bank
        smi += widx*32;         // moves the index inside the bank for the correct word

        bankPointer[smi] = frPointer[widx];
        // __syncthreads();
    }
}

__device__ void rsm_fr(unsigned index, fr_t &output){
    //considers that shared mem pointer is called fr_roots_sm
    uint32_t *bankPointer;
    uint32_t *frPointer;
    bankPointer = (uint32_t*) fr_roots_sm;
    frPointer=(uint32_t*) output;

    //index changing: similar to aes shiftcolumns, on 32 columns
    unsigned rotN = index/32; //rotation amount
    index = ((index + 32 - rotN)%32)+ (rotN*32);

    int smi; //index of the shared memory (as uint32_t)

    for(int widx=0; widx<sizeof(fr_t)/4; widx++){ //8 words
        smi = (index/32)*32*8; // puts the index bank[0][0] for this warp
        smi += index%32;        // moves the index foward to the correct bank
        smi += widx*32;         // moves the index inside the bank for the correct word

        frPointer[widx] = bankPointer[smi];
        // __syncthreads();
    }
}

#endif
