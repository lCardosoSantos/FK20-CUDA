#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022 Dag Arne Osvik

#include "fr.cuh"
#include "fp.cuh"
#include "g1.cuh"
#include "fk20.cuh"
#include "fk20test.cuh"

//debug macros for dumping elements to file

#define WRITEU64(writing_stream, var, nu64Elem) do{ \
    uint64_t *pointer = (uint64_t *)(*var); \
    for (int count=0; count<(nu64Elem); count++){ \
        fprintf(writing_stream,"%016lx\n",pointer[count]); \
    } \
}while(0)

#define WRITEU64TOFILE(fileName, var, nu64Elem) do{ \
    FILE * filepointer = fopen(fileName, "w");     \
    WRITEU64(filepointer, var, (nu64Elem));           \
    fclose(filepointer);                           \
}while(0) 

#define WRITEU64STDOUT(var, nu64Elem) do{ \
    uint64_t *pointer = (uint64_t *)(*var); \
    for (int count=0; count<(nu64Elem); count++){ \
        printf("%016lx\n",pointer[count]); \
    } \
}while(0)

// Testvector inputs

extern __managed__ g1p_t xext_fft[16][512];
extern __managed__ fr_t polynomial[512*4096];

// Intermediate values

extern __managed__ fr_t toeplitz_coefficients[512*16][512];
extern __managed__ fr_t toeplitz_coefficients_fft[512*16][512];
extern __managed__ g1p_t hext_fft[512*512];
extern __managed__ g1p_t h[512*512];

// Testvector output

extern __managed__ g1p_t h_fft[512*512];

// Workspace

static __managed__ uint8_t cmp[512*16*512];
static __managed__ fr_t fr_tmp_[512*16*512];
static __managed__ g1p_t g1p_tmp[512*512];

//512 tests
void toeplitz_coefficients2toeplitz_coefficients_fft_512();
void h2h_fft_512();
void h_fft2h_512();
void hext_fft2h_512();

void fk20_poly2toeplitz_coefficients_512(int execN);
void fk20_poly2hext_fft_512();
void fk20_poly2h_fft_512();

int main() {
    /*
    //all tests
    toeplitz_coefficients2toeplitz_coefficients_fft_512();
    h2h_fft_512();
    h_fft2h_512();
    hext_fft2h_512();
    fk20_poly2toeplitz_coefficients_512(); //problematic one
    fk20_poly2hext_fft_512();
    fk20_poly2h_fft_512();
    */

    //remove uncertainty
    for(int i=0; i<(512*16*512); i++){
        fr_tmp_[i][0]=1;
        fr_tmp_[i][1]=1;
        fr_tmp_[i][2]=1;
        fr_tmp_[i][3]=1;
    } 
    
    fk20_poly2toeplitz_coefficients_512(0); //problematic one
    toeplitz_coefficients2toeplitz_coefficients_fft_512();
    fk20_poly2toeplitz_coefficients_512(1); //problematic one

    return 0;
}

/*
Luan's notes
causes fk20_poly2toeplitz_coefficients: polynomial -> toeplitz_coefficients to fail idx0201:
    toeplitz_coefficients -> toeplitz_coefficients_fft 

causes fk20_poly2h_fft: polynomial -> h_fft to fail (hipErrorIllegalAddress)
    fr_fft: toeplitz_coefficients -> toeplitz_coefficients_fft

    g1p_fft: h -> h_fft

    g1p_ift: h_fft -> h

    g1p_ift: hext_fft -> h

    fk20_poly2toeplitz_coefficients: polynomial -> toeplitz_coefficients

    fk20_poly2hext_fft: polynomial -> hext_fft


Some awk magix
awk 'getline p<f && p!=$0 {print "Line " NR ":" RS $0 RS p; exit}' f=file2 file1
*/

void toeplitz_coefficients2toeplitz_coefficients_fft_512(){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    printf("=== RUN   %s\n", "fr_fft: toeplitz_coefficients -> toeplitz_coefficients_fft");
    start = clock();
    fr_fft_wrapper<<<512*16, 256, fr_sharedmem>>>(fr_tmp_, (fr_t *)toeplitz_coefficients);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fr_fft_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*16*512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "fr_eq_wrapper", cmp, 512, fr_tmp_, h_fft); fflush(stdout);

    fr_eq_wrapper<<<256, 32>>>(cmp, 512*16*512, fr_tmp_, (fr_t *)toeplitz_coefficients_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error fr_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check FFT result

    for (int i=0; pass && i<512*16*512; i++)
        if (cmp[i] != 1) {
            printf("FFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}

void h2h_fft_512(){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_fft_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));


    printf("=== RUN   %s\n", "g1p_fft: h -> h_fft");
    start = clock();
    g1p_fft_wrapper<<<512, 256, g1p_sharedmem>>>(g1p_tmp, h);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess) printf("Error g1p_fft_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Clear comparison results

    for (int i=0; i<512*512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h_fft); fflush(stdout);

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512*512, g1p_tmp, h_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Check FFT result

    for (int i=0; pass && i<512*512; i++)
        if (cmp[i] != 1) {
            printf("FFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);

}

void h_fft2h_512(){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_ift_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));


    printf("=== RUN   %s\n", "g1p_ift: h_fft -> h");

    start = clock();
    g1p_ift_wrapper<<<512, 256, g1p_sharedmem>>>(g1p_tmp, h_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error g1p_ift_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512*512, g1p_tmp, h); fflush(stdout);

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512*512, g1p_tmp, h);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check IFT result

    for (int i=0; pass && i<512*512; i++)
        if (cmp[i] != 1) {
            printf("IFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);

}

void hext_fft2h_512(){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_ift_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "g1p_ift: hext_fft -> h");

    start = clock();
    g1p_ift_wrapper<<<1, 256, g1p_sharedmem>>>(g1p_tmp, hext_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error g1p_ift_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h); fflush(stdout);

    g1p_eq_wrapper<<<8, 32>>>(cmp, 256, g1p_tmp, h);    // Note: h, not hext, hence 256, not 512

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check IFT result

    for (int i=0; pass && i<256; i++)
        if (cmp[i] != 1) {
            printf("IFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);

}

void fk20_poly2toeplitz_coefficients_512(int execN){ //TODO: Luan main work focus
        char polyFilename [64];
        char fr_tmpFilename [64];
        char toeplitzFilename [64];
        //remove some uncertainty
        //memset(fr_tmp_, 1, 512*16*512*sizeof(fr_t)); //fr_tmp_[512*16*512];
        //for(int i=0; i<(512*16*512); i++){
        //    fr_tmp_[i][0]=0;//1;
        //    fr_tmp_[i][1]=0;//1;
        //    fr_tmp_[i][2]=0;//1;
        //    fr_tmp_[i][3]=0;//1;
        //} 


    hipError_t err;
    bool pass = true;
    clock_t start, end;

    printf("=== RUN   %s\n", "fk20_poly2toeplitz_coefficients: polynomial -> toeplitz_coefficients");
    start = clock();
        //sprintf(polyFilename,     "pol%d-%d.out", execN, 0  );
        //sprintf(fr_tmpFilename,   "tmp%d-%d.out", execN, 0  );
        //sprintf(toeplitzFilename, "toe%d-%d.out", execN, 0  );
        //WRITEU64TOFILE(polyFilename,     polynomial,            512*4096*4);
        //WRITEU64TOFILE(fr_tmpFilename,   fr_tmp_,                512*16*512*4);
        //WRITEU64TOFILE(toeplitzFilename, toeplitz_coefficients, 512*16*512*4);
    fk20_poly2toeplitz_coefficients<<<512, 256>>>(fr_tmp_, polynomial);
    err = hipDeviceSynchronize();
        //sprintf(polyFilename,     "pol%d-%d.out", execN, 1  );
        //sprintf(fr_tmpFilename,   "tmp%d-%d.out", execN, 1  );
        //sprintf(toeplitzFilename, "toe%d-%d.out", execN, 1  );
        //WRITEU64TOFILE(polyFilename,     polynomial,            512*4096*4);
        //WRITEU64TOFILE(fr_tmpFilename,   fr_tmp_,                512*16*512*4);
        //WRITEU64TOFILE(toeplitzFilename, toeplitz_coefficients, 512*16*512*4);

    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2toeplitz_coefficients: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*16*512; i++)
        cmp[i] = 0;

    fr_eq_wrapper<<<1, 32>>>(cmp, 512*16*512, fr_tmp_, (fr_t *)toeplitz_coefficients);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error fr_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result
    
    for (int i=0; pass && i<512*16*512; i++)
        if (cmp[i] != 1) {
            printf("poly2toeplitz_coefficients error at idx 0x%04x\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}

void fk20_poly2hext_fft_512(){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    pass = true;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(fk20_poly2hext_fft), hipFuncAttributeMaxDynamicSharedMemorySize, fr_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "fk20_poly2hext_fft: polynomial -> hext_fft");

    start = clock();
    fk20_poly2hext_fft<<<512, 256, g1p_sharedmem>>>(g1p_tmp, polynomial, (const g1p_t *)xext_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2hext_fft: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<1, 32>>>(cmp, 512*512, g1p_tmp, (g1p_t *)hext_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<512*512; i++)
        if (cmp[i] != 1) {
            pass = false;
        }

    PRINTPASS(pass);

}

void fk20_poly2h_fft_512(){
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(fk20_poly2h_fft), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "fk20_poly2h_fft: polynomial -> h_fft");

    start = clock();
    fk20_poly2h_fft<<<512, 256, g1p_sharedmem>>>(g1p_tmp, polynomial, (const g1p_t *)xext_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2h_fft: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<1, 32>>>(cmp, 512*512, g1p_tmp, (g1p_t *)h_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<512*512; i++)
        if (cmp[i] != 1) {
            pass = false;
        }

    PRINTPASS(pass);
}



// vim: ts=4 et sw=4 si
