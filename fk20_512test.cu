#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022 Dag Arne Osvik
#include <cstring>
#include<time.h>
#include "fr.cuh"
#include "fp.cuh"
#include "g1.cuh"
#include "fk20.cuh"
#include "fk20test.cuh"


// Testvector inputs

extern __managed__ g1p_t xext_fft[16][512];
extern __managed__ fr_t polynomial[512*4096];

// Intermediate values

extern __managed__ fr_t toeplitz_coefficients[512*16][512];
extern __managed__ fr_t toeplitz_coefficients_fft[512*16][512];
extern __managed__ g1p_t hext_fft[512*512];
extern __managed__ g1p_t h[512*512];

// Testvector output

extern __managed__ g1p_t h_fft[512*512];

// Workspace

static __managed__ uint8_t cmp[512*16*512];
static __managed__ fr_t fr_tmp_[512*16*512];
static __managed__ g1p_t g1p_tmp[512*512];

#define PatternOnWorkspaceMemory
#ifdef PatternOnWorkspaceMemory
    #define PTRN_G1PTMP memset(g1p_tmp, 0x88, 512*512*sizeof(g1p_t));
    #define PTRN_FRTMP  memset(fr_tmp_, 0x88, 512*16*512*sizeof(fr_t));
#else
    #define PTRN_G1PTMP 
    #define PTRN_FRTMP  
#endif

//512 tests
void toeplitz_coefficients2toeplitz_coefficients_fft_512(unsigned rows);
void h2h_fft_512(unsigned rows);
void h_fft2h_512(unsigned rows);
void hext_fft2h_512(unsigned rows);
void hext_fft2h_fft_512(unsigned rows);

void fk20_poly2toeplitz_coefficients_512(unsigned rows);
void fk20_poly2hext_fft_512(unsigned rows);
void fk20_poly2h_fft_512(unsigned rows);
void fk20_msmloop_512(unsigned rows);
void fk20_poly2toeplitz_coefficients_fft_test(unsigned rows);

int main(int argc, char **argv) {

    unsigned rows = 2;

    if (argc > 1)
        rows = atoi(argv[1]);

        if (rows < 1)
            rows = 1;

        if (rows > 512)
            rows = 512;
    
    //all tests
    toeplitz_coefficients2toeplitz_coefficients_fft_512(rows);
    h2h_fft_512(rows);
    h_fft2h_512(rows); 
    hext_fft2h_512(rows);
    //hext_fft2h_fft_512(rows); //fails, but components work
    fk20_poly2toeplitz_coefficients_512(rows); //TODO: parameter is debug, remove.
    fk20_poly2hext_fft_512(rows); 
    fk20_msmloop_512(rows);

    fk20_poly2h_fft_512(rows);
    
    //fk20_poly2toeplitz_coefficients_fft_test(rows); //TODO: Superfluos function?

    return 0;
}

void toeplitz_coefficients2toeplitz_coefficients_fft_512(unsigned rows){
    PTRN_FRTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    printf("=== RUN   %s\n", "fr_fft: toeplitz_coefficients -> toeplitz_coefficients_fft");
    start = clock();
    fr_fft_wrapper<<<rows*16, 256, fr_sharedmem>>>(fr_tmp_, (fr_t *)toeplitz_coefficients);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fr_fft_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<rows*16*512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "fr_eq_wrapper", cmp, 512, fr_tmp_, h_fft); fflush(stdout);

    fr_eq_wrapper<<<256, 32>>>(cmp, rows*16*512, fr_tmp_, (fr_t *)toeplitz_coefficients_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error fr_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check FFT result

    for (int i=0; pass && i<rows*16*512; i++)
        if (cmp[i] != 1) {
            printf("FFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}

void h2h_fft_512(unsigned rows){
    PTRN_G1PTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_fft_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));


    printf("=== RUN   %s\n", "g1p_fft: h -> h_fft");
    start = clock();
    g1p_fft_wrapper<<<rows, 256, g1p_sharedmem>>>(g1p_tmp, h);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess) printf("Error g1p_fft_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Clear comparison results

    for (int i=0; i<rows*512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h_fft); fflush(stdout);

    g1p_eq_wrapper<<<16, 32>>>(cmp, rows*512, g1p_tmp, h_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Check FFT result

    for (int i=0; pass && i<rows*512; i++)
        if (cmp[i] != 1) {
            printf("FFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);

}

void h_fft2h_512(unsigned rows){
    PTRN_G1PTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_ift_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));


    printf("=== RUN   %s\n", "g1p_ift: h_fft -> h");

    start = clock();
    g1p_ift_wrapper<<<512, 256, g1p_sharedmem>>>(g1p_tmp, h_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error g1p_ift_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512*512, g1p_tmp, h); fflush(stdout);

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512*512, g1p_tmp, h);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check IFT result

    for (int i=0; pass && i<512*512; i++)
        if (cmp[i] != 1) {
            printf("IFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);

}

void hext_fft2h_512(unsigned rows){
    PTRN_G1PTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(g1p_ift_wrapper), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "g1p_ift: hext_fft -> h");

    start = clock();
    g1p_ift_wrapper<<<rows, 256, g1p_sharedmem>>>(g1p_tmp, hext_fft);
    fk20_hext2h<<<rows, 256>>>(g1p_tmp);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error g1p_ift_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    // printf("  %s(%p, %d, %p, %p)\n", "g1p_eq_wrapper", cmp, 512, g1p_tmp, h); fflush(stdout);

    g1p_eq_wrapper<<<8, 32>>>(cmp, rows*512, g1p_tmp, h);   

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    // Check IFT result

    for (int i=0; pass && i<rows*256; i++)
        if (cmp[i] != 1) {
            printf("IFT error %d\n", i);
            pass = false;
        }

    PRINTPASS(pass);

}

void fk20_poly2toeplitz_coefficients_512(unsigned rows){ 
    PTRN_FRTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    printf("=== RUN   %s\n", "fk20_poly2toeplitz_coefficients: polynomial -> toeplitz_coefficients");
    start = clock();

    fk20_poly2toeplitz_coefficients<<<rows, 256, fr_sharedmem>>>(fr_tmp_, polynomial);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2toeplitz_coefficients: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*16*512; i++)
        cmp[i] = 0;

    fr_eq_wrapper<<<1, 32>>>(cmp, rows*16*512, fr_tmp_, (fr_t *)toeplitz_coefficients);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error fr_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result
    
    for (int i=0; pass && i<rows*16*512; i++)
        if (cmp[i] != 1) {
            printf("poly2toeplitz_coefficients error at idx 0x%04x\n", i);
            pass = false;
        }

    PRINTPASS(pass);
}

void fk20_poly2hext_fft_512(unsigned rows){
    PTRN_G1PTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    pass = true;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(fk20_poly2hext_fft), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "fk20_poly2hext_fft: polynomial -> hext_fft");

    start = clock();
    fk20_poly2hext_fft<<<rows, 256, g1p_sharedmem>>>(g1p_tmp, polynomial, (const g1p_t *)xext_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2hext_fft: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));
    // Clear comparison results

    for (int i=0; i<512*512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<1, 32>>>(cmp, rows*512, g1p_tmp, (g1p_t *)hext_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<rows*512; i++)
        if (cmp[i] != 1) {
            pass = false;
            printf("Error at idx %d...\n", i);
            break;
        }

    PRINTPASS(pass);
}

void fk20_poly2h_fft_512(unsigned rows){
    PTRN_G1PTMP; PTRN_FRTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    printf("=== RUN   %s\n", "fk20_poly2h_fft: polynomial -> h_fft");

    start = clock();
    fk20_poly2h_fft(g1p_tmp, polynomial, (const g1p_t *)xext_fft, rows);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2h_fft: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<rows*512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<1, 32>>>(cmp, rows*512, g1p_tmp, (g1p_t *)h_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<rows*512; i++)
        if (cmp[i] != 1) {
            pass = false;
        }

    PRINTPASS(pass);
}

void hext_fft2h_fft_512(unsigned rows){
    // TODO: Superfluous test?
    // Note from u1d4db:    I think we can remove this function, since it is just ift + zerohal + fft
    //                      it is also probably broken with recent code changes.
    PTRN_G1PTMP;
    hipError_t err;
    bool pass = true;
    clock_t start, end;

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(fk20_hext_fft2h_fft), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    printf("=== RUN   %s\n", "hext_fft2h_fft_512: hext_fft -> h_fft");

    start = clock();
    fk20_hext_fft2h_fft<<<rows, 256, g1p_sharedmem>>>(g1p_tmp, hext_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_hext_fft2h_fft: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<8, 32>>>(cmp, rows*512, g1p_tmp, h);   

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    for (int i=0; pass && i<rows*256; i++)
        if (cmp[i] != 1) {
            printf("fk20_hext_fft2h_fft error %d...\n", i);
            pass = false;
            break;
        }

    PRINTPASS(pass);

}

void fk20_msmloop_512(unsigned rows){
    clock_t start, end;
    hipError_t err;
    bool pass = true;

    printf("=== RUN   %s\n", "fk20_msm: Toeplitz_coefficients+xext_fft -> hext_fft");
    start = clock();
    
    fk20_msm<<<rows, 256>>>(g1p_tmp, (const fr_t*)toeplitz_coefficients_fft, (const g1p_t*)xext_fft);

    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_msm: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<rows*512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<16, 32>>>(cmp, rows*512, g1p_tmp, (g1p_t *)hext_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<rows*512; i++)
        if (cmp[i] != 1) {
            pass = false;
            printf("Fails at idx %d", i);
            break;
        }

    PRINTPASS(pass);
}

void fk20_poly2toeplitz_coefficients_fft_test(unsigned rows){
    // TODO: Superfluous test?
    // Note from u1d4db:    I think we can remove this function, since it is just poly2tc + fr_fft
    //                      it is also probably broken with recent code changes.
    PTRN_FRTMP;
    clock_t start, end;
    hipError_t err;
    bool pass = true;

    printf("=== RUN   %s\n", "fk20_poly2toeplitz_coefficients_fft: polynomial -> toeplitz_coefficients_fft");
    memset(fr_tmp_, 0xdeadbeef,512*16*512*sizeof(fr_t)); //pattern on tmp dest.
    start = clock();
    fk20_poly2toeplitz_coefficients_fft<<<rows, 256>>>(fr_tmp_, polynomial);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2toeplitz_coefficients_fft: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512*16*512; i++)
        cmp[i] = 0;

    fr_eq_wrapper<<<16, 256>>>(cmp, rows*16*512, fr_tmp_, (fr_t *)toeplitz_coefficients_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error fr_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<rows*16*512; i++)
        if (cmp[i] != 1) {
            printf("poly2tc error %04x\n", i);
            pass = false;
            break;
        }

    PRINTPASS(pass);
}


// vim: ts=4 et sw=4 si
