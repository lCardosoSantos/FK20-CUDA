#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022 Dag Arne Osvik

#include "fp.cuh"
#include "fp_add.cuh"

__device__ void fp_add(fp_t &z, const fp_t &x, const fp_t &y) {
    uint64_t
        x0 = x[0], y0 = y[0], z0,
        x1 = x[1], y1 = y[1], z1,
        x2 = x[2], y2 = y[2], z2,
        x3 = x[3], y3 = y[3], z3,
        x4 = x[4], y4 = y[4], z4,
        x5 = x[5], y5 = y[5], z5;

    asm volatile (
    "\n\t{"
    "\n\t.reg .u64 z<6>, x<6>, y<6>;"
    "\n\t.reg .u32 t6;"
    "\n\t.reg .pred cp;"

    "\n\tmov.u64 x0,  %6;"
    "\n\tmov.u64 x1,  %7;"
    "\n\tmov.u64 x2,  %8;"
    "\n\tmov.u64 x3,  %9;"
    "\n\tmov.u64 x4, %10;"
    "\n\tmov.u64 x5, %11;"

    "\n\tmov.u64 y0, %12;"
    "\n\tmov.u64 y1, %13;"
    "\n\tmov.u64 y2, %14;"
    "\n\tmov.u64 y3, %15;"
    "\n\tmov.u64 y4, %16;"
    "\n\tmov.u64 y5, %17;"

FP_ADD(z, x, y)

    "\n\tmov.u64 %0,  z0;"
    "\n\tmov.u64 %1,  z1;"
    "\n\tmov.u64 %2,  z2;"
    "\n\tmov.u64 %3,  z3;"
    "\n\tmov.u64 %4,  z4;"
    "\n\tmov.u64 %5,  z5;"

    "\n\t}"
    :
    "=l"(z0), "=l"(z1), "=l"(z2), "=l"(z3), "=l"(z4), "=l"(z5)
    :
    "l"(x0), "l"(x1), "l"(x2), "l"(x3), "l"(x4), "l"(x5),
    "l"(y0), "l"(y1), "l"(y2), "l"(y3), "l"(y4), "l"(y5)
    ); 

    z[0] = z0, z[1] = z1, z[2] = z2, z[3] = z3, z[4] = z4, z[5] = z5;
}

// vim: ts=4 et sw=4 si
