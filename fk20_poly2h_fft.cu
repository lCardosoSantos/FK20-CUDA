#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#include "fr.cuh"
#include "g1.cuh"
#include "fk20.cuh"

#include "fk20test.cuh"
#include "fk20_testvector.cuh"

#define ROWS 512

#define SET_SHAREDMEM(SZ, FN) \
    err = hipFuncSetAttribute(reinterpret_cast<const void*>(FN), hipFuncAttributeMaxDynamicSharedMemorySize, SZ); \
    hipDeviceSynchronize(); \
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

#define CUDASYNC     err = hipDeviceSynchronize(); \
                     if (err != hipSuccess) printf("Error: %d (%s)\n", err, hipGetErrorName(err))

static __managed__ fr_t fr[ROWS*16*512]; // 256 KiB per threadblock
static __managed__ g1p_t g1p[ROWS*512];  // 72 KiB per threadblock

////////////////////////////////////////////////////////////////////////////////

__global__ void fk20_hext2h(g1p_t *h) {
    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number

    h += 512*bid;
    g1p_inf(h[256+tid]);
}

////////////////////////////////////////////////////////////////////////////////
// fk20_poly2h_fft(): polynomial + xext_fft -> h_fft
// This is the full execution of FK20. 
// parameters:
// - in  xext_fft   array with 16*512 elements, computed using fk20_setup2xext_fft()
// - in  polynomial array with 16*512*rows elements
// - in  rows       number of rows to process in one kernel launch
// - out h_fft      array with    512*rows elements

__host__ void fk20_poly2h_fft(g1p_t *h_fft, const fr_t *polynomial, const g1p_t xext_fft[8192], unsigned rows) {
    hipError_t err;
    clock_t start, end;

    // Setup

    SET_SHAREDMEM(fr_sharedmem,  fr_fft_wrapper);
    SET_SHAREDMEM(fr_sharedmem,  fk20_msm);
    SET_SHAREDMEM(g1p_sharedmem, g1p_fft_wrapper);
    SET_SHAREDMEM(g1p_sharedmem, g1p_ift_wrapper);

    // polynomial -> tc
    printf("polynomial -> tc\n"); fflush(stdout);

    start = clock();
    fk20_poly2toeplitz_coefficients<<<rows, 256, fr_sharedmem>>>(fr, polynomial);
    CUDASYNC;
    end = clock();
    printf(" (%.1f ms)\n", (end - start) * (1000. / CLOCKS_PER_SEC));

    // tc -> tc_fft
    printf("tc -> tc_fft\n"); fflush(stdout);
    start = clock();
    for(int i=0; i<16; i++){
        fr_fft_wrapper<<<rows, 256, fr_sharedmem>>>(fr+512*i, fr+512*i);
    }

    CUDASYNC;
    end = clock();
    printf(" (%.1f ms)\n", (end - start) * (1000. / CLOCKS_PER_SEC));

    // tc_fft -> hext_fft
    printf("tc_fft -> hext_fft\n"); fflush(stdout);

    start = clock();
    fk20_msm<<<rows, 256>>>(g1p, fr, xext_fft);
    CUDASYNC;
    end = clock();
    printf(" (%.1f ms)\n", (end - start) * (1000. / CLOCKS_PER_SEC));

    // hext_fft -> hext
    printf("hext_fft -> hext\n"); fflush(stdout);
    start = clock();
    g1p_ift_wrapper<<<rows, 256, g1p_sharedmem>>>(g1p, g1p);

    CUDASYNC;
    end = clock();
    printf(" (%.1f ms)\n", (end - start) * (1000. / CLOCKS_PER_SEC));

    // hext -> h
    printf("hext -> h\n"); fflush(stdout);

    start = clock();
    fk20_hext2h<<<rows, 256>>>(g1p);
    CUDASYNC;
    end = clock();
    printf(" (%.1f ms)\n", (end - start) * (1000. / CLOCKS_PER_SEC));

    // h -> h_fft
    printf("h -> h_fft\n"); fflush(stdout);

    start = clock();
    g1p_fft_wrapper<<<rows, 256, g1p_sharedmem>>>(g1p, g1p);
    CUDASYNC;
    end = clock();
    printf(" (%.1f ms)\n", (end - start) * (1000. / CLOCKS_PER_SEC));
}

// vim: ts=4 et sw=4 si
