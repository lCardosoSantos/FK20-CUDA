#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022 Dag Arne Osvik

#include <stdio.h>

#include "g1.cuh"
#include "fk20.cuh"
#include "fk20test.cuh"
#include "fk20_testvector.cuh"

static __managed__ uint8_t cmp[16*512];
static __managed__ fr_t fr_tmp[16*512];
static __managed__ g1p_t g1p_tmp[512];

void FK20TestPoly() {

    hipError_t err;
    bool pass = true;
    clock_t start, end;

    //////////////////////////////////////////////////

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(fk20_poly2hext_fft), hipFuncAttributeMaxDynamicSharedMemorySize, fr_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(fk20_poly2h_fft), hipFuncAttributeMaxDynamicSharedMemorySize, g1p_sharedmem);
    hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error hipFuncSetAttribute: %s:%d, error %d (%s)\n", __FILE__, __LINE__, err, hipGetErrorName(err));

    //////////////////////////////////////////////////

    printf("=== RUN   %s\n", "fk20_poly2toeplitz_coefficients: polynomial -> toeplitz_coefficients");

    start = clock();
    fk20_poly2toeplitz_coefficients<<<1, 256>>>(fr_tmp, polynomial);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2toeplitz_coefficients: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<16*512; i++)
        cmp[i] = 0;

    fr_eq_wrapper<<<256, 32>>>(cmp, 16*512, fr_tmp, (fr_t *)toeplitz_coefficients);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error fr_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<16*512; i++)
        if (cmp[i] != 1) {
            printf("poly2tc error %04x\n", i);
            pass = false;
        }

    printf("--- %s\n", pass ? "PASS" : "FAIL");

    //////////////////////////////////////////////////

    pass = true;

    printf("=== RUN   %s\n", "fk20_poly2hext_fft: polynomial -> hext_fft");

    start = clock();
    fk20_poly2hext_fft<<<1, 256, fr_sharedmem>>>(g1p_tmp, polynomial, (const g1p_t *)xext_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2hext_fft: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512, g1p_tmp, (g1p_t *)hext_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<512; i++)
        if (cmp[i] != 1) {
            pass = false;
        }

    printf("--- %s\n", pass ? "PASS" : "FAIL");

    //////////////////////////////////////////////////

    pass = true;


    printf("=== RUN   %s\n", "fk20_poly2h_fft: polynomial -> h_fft");

    start = clock();
    fk20_poly2h_fft<<<1, 256, g1p_sharedmem>>>(g1p_tmp, polynomial, (const g1p_t *)xext_fft);
    err = hipDeviceSynchronize();
    end = clock();

    if (err != hipSuccess)
        printf("Error fk20_poly2h_fft: %d (%s)\n", err, hipGetErrorName(err));
    else
        printf(" (%.3f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

    // Clear comparison results

    for (int i=0; i<512; i++)
        cmp[i] = 0;

    g1p_eq_wrapper<<<16, 32>>>(cmp, 512, g1p_tmp, (g1p_t *)h_fft);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Error g1p_eq_wrapper: %d (%s)\n", err, hipGetErrorName(err));

    // Check result

    for (int i=0; i<512; i++)
        if (cmp[i] != 1) {
            pass = false;
        }

    printf("--- %s\n", pass ? "PASS" : "FAIL");

    //////////////////////////////////////////////////
}

// vim: ts=4 et sw=4 si
