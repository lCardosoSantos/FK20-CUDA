#include "hip/hip_runtime.h"
#include <stdio.h>

#include "fr.cuh"
#include "fk20.cuh"

// Workspace in shared memory

extern __shared__ fr_t fr[];

// FFT over Fr
// input and output may freely overlap

__device__ void fr_fft(fr_t *output, const fr_t *input) {
    // One FFT of size 512 elements per thread block
    // Must be called with 256threads per block
    // No interleaving of data for different FFTs

    if (gridDim.y  !=   1) return;
    if (gridDim.z  !=   1) return;
    if (blockDim.x != 256) return;
    if (blockDim.y !=   1) return;
    if (blockDim.z !=   1) return;

    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number
    unsigned l, r, w, src, dst;

    // Adjust IO pointers to point at each thread block's data

    input  += 512*bid;
    output += 512*bid;

    // Copy inputs to workspace

    src = tid;
    // dst = 9 last bits of src reversed
    asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(dst) : "r"(src << (32-9)));

    fr_cpy(fr[dst], input[src]);

    src |= 256;
    dst |= 1;

    fr_cpy(fr[dst], input[src]);

    __syncthreads();

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    //fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 1

    w = (tid & 1) << 7;
    l = tid + (tid & -2U);
    r = l | 2;

    if (w) fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;

    fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    //// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    fr_mul(fr[r], fr_roots[w]);
    fr_addsub(fr[l], fr[r]);

    __syncthreads();

    // Copy results to output, no shuffle

    src = tid;
    dst = src;

    fr_cpy(output[dst], fr[src]);

    src += 256;
    dst += 256;

    fr_cpy(output[dst], fr[src]);
}

// Inverse FFT over Fr

__device__ void fr_ift(fr_t *output, const fr_t *input) {
    // One inverse FFT of size 512 per thread block
    // No interleaving of data for different FFTs

    if (gridDim.y  !=   1) return;
    if (gridDim.z  !=   1) return;
    if (blockDim.x != 256) return;
    if (blockDim.y !=   1) return;
    if (blockDim.z !=   1) return;

    unsigned tid = threadIdx.x; // Thread number
    unsigned bid = blockIdx.x;  // Block number
    unsigned l, r, w, src, dst;

    // Adjust IO pointers to point at each thread block's data

    input  += 512*bid;
    output += 512*bid;

    // Copy inputs to workspace, no shuffle

    src = tid;
    dst = src;

    fr_cpy(fr[dst], input[src]);

    src += 256;
    dst += 256;

    fr_cpy(fr[dst], input[src]);

    __syncthreads();

    //// Stage 8

    w = (tid & 255) << 0;
    l = tid + (tid & -256U);
    r = l | 256;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 7

    w = (tid & 127) << 1;
    l = tid + (tid & -128U);
    r = l | 128;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 6

    w = (tid & 63) << 2;
    l = tid + (tid & -64U);
    r = l | 64;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 5

    w = (tid & 31) << 3;
    l = tid + (tid & -32U);
    r = l | 32;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 4

    w = (tid & 15) << 4;
    l = tid + (tid & -16U);
    r = l | 16;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 3

    w = (tid & 7) << 5;
    l = tid + (tid & -8U);
    r = l | 8;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 2

    w = (tid & 3) << 6;
    l = tid + (tid & -4U);
    r = l | 4;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    //// Stage 1

    w = (tid & 1) << 0;
    l = tid + (tid & -2U);
    r = l | 2;

    fr_addsub(fr[l], fr[r]);
    fr_mul(fr[l], fr_roots[513]);      // 2**-9
    fr_mul(fr[r], fr_roots[513+w]);    // w ? 2**-9/fr_roots[128] : 2**-9

    __syncthreads();

    //// Stage 0

    w = 0;
    l = 2 * tid;
    r = l | 1;

    fr_addsub(fr[l], fr[r]);
    //fr_mul(fr[r], fr_roots[512-w]);

    __syncthreads();

    // Copy results to output

    dst = tid;
    // src = 9 last bits of dst reversed
    asm volatile ("\n\tbrev.b32 %0, %1;" : "=r"(src) : "r"(dst << (32-9)));

    fr_cpy(output[dst], fr[src]);

    dst |= 256;
    src |= 1;

    fr_cpy(output[dst], fr[src]);
}

// Kernel wrappers for device-side FFT functions

__global__ void fr_fft_wrapper(fr_t *output, const fr_t *input) { fr_fft(output, input); }
__global__ void fr_ift_wrapper(fr_t *output, const fr_t *input) { fr_ift(output, input); }

// vim: ts=4 et sw=4 si
