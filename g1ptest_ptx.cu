#include "hip/hip_runtime.h"
// bls12_381: Arithmetic for BLS12-381
// Copyright 2022-2023 Dag Arne Osvik
// Copyright 2022-2023 Luan Cardoso dos Santos

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include "g1.cuh"
#include "test.h"
// #include "g1test.cuh"

#include "g1p_ptx.cuh"

#define TESTVALS 256

typedef struct {
    uint64_t val[22];
} testval_t;

__managed__ testval_t testval[TESTVALS];

////////////////////////////////////////////////////////////

/**
 * @brief initialization
 * 
 */
void init() {

    printf("%s\n", __func__);

    testinit();

    /*
    uint64_t t[2*TESTVALS];

    FILE *pf = fopen("/dev/urandom", "r");

    if (!pf)
        return;

    size_t result = fread(&testval[i], sizeof(testval_t), TESTVALS-i, pf);
    */
}

////////////////////////////////////////////////////////////
//shortcut for kernel declaration
__global__ void G1_ADD_PTX(testval_t *testval);
__global__ void G1_SUB_PTX(testval_t *testval);
__global__ void G1_DBL_PTX(testval_t *testval);
__global__ void G1_ADDSUB_PTX(testval_t *testval);


//Shorthand for testing a function, with an error check and timer
#define TEST(X) \
    start = clock(); \
    X <<<grid,block>>> (&testval[0]); \
    err = hipDeviceSynchronize(); \
    end = clock(); \
    if (err != hipSuccess) printf("Error %d (%s)\n", err, hipGetErrorName(err)); \
    printf(" (%.2f s)\n", (end - start) * (1.0 / CLOCKS_PER_SEC));

////////////////////////////////////////////////////////////

/**
 * @brief Test for points in G1
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
int main(int argc, char **argv) {
    clock_t start, end;
    hipError_t err;
#if 1
    dim3 block(1,1,1);
    dim3 grid(1,1,1);
#else
    dim3 block(32,8,1);
    dim3 grid(82,1,1);
#endif

    unsigned rows = 2;

    if (argc > 1)
        rows = atoi(argv[1]);

    if (rows > 512)
        rows = 512;

    init();

    TEST(G1_ADD_PTX);
    TEST(G1_SUB_PTX);
    TEST(G1_DBL_PTX);
    TEST(G1_ADDSUB_PTX);


    return err;
}

__global__ void G1_ADD_PTX(testval_t *testval){
    printf("== TEST %s NOT IMPLEMENTED\n", __func__);

}

__global__ void G1_SUB_PTX(testval_t *testval){
    printf("== TEST %s NOT IMPLEMENTED\n", __func__);

}

__global__ void G1_DBL_PTX(testval_t *testval){
    printf("== TEST %s \n", __func__);
    bool pass = true;
    size_t count = 0;

    g1p_t out1, out0, in1, in0;
    g1p_t p, q, u, v;

    g1p_gen(p);
    g1p_gen(in0);

    g1p_print("cuda p = ", p);
    g1p_print("ptxm in0= ", in0); 
    printf("\n\n");

    for (int i=0; pass&i<20000; i++){
        g1p_dbl(p); 
        g1m(OP_DBL, out0, in0, in0, in0);

        if(g1p_neq(p, in0)){
            pass = false;
            printf("%d: FAILED\n", i);
            printf("FAILED\n" );
            g1p_print("cuda = ", p);
            g1p_print("ptxm = ", in0);
            pass = false;
        }

        //g1p_cpy(in0, out0);
        ++count;
    }

    if (!pass || (blockIdx.x | blockIdx.y | blockIdx.z | threadIdx.x | threadIdx.y | threadIdx.z) == 0){
        printf("%ld tests\n", count);
        PRINTPASS(pass);
    }

    }

__global__ void G1_ADDSUB_PTX(testval_t *testval){
    printf("== TEST %s NOT IMPLEMENTED\n", __func__);

}
